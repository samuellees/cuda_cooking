#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hipblas.h>
#include "stdio.h"
#include "gemv.h"

#ifndef CUDA_CHECK

#define CUDA_CHECK(code)                                                  \
  {                                                                       \
    hipError_t status = (code);                                         \
    if ((status) != hipSuccess) {                                          \
      fprintf(stderr, "CUDA error in file: %s, line: %d, %s\n", __FILE__, \
              __LINE__, hipGetErrorString((status)));                      \
      exit((status));                                                       \
    }                                                                     \
  }
#endif

static const int threadsPerBlock = 128;
static const int blocksPerGrid = 32;

__global__ void kernel_naive(const Matrix A, const Vector X, Vector Y) {
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = tid; i < A.n_row; i += threadsPerBlock * blocksPerGrid) {
    float temp = 0;
    for (int j = 0; j < A.n_col; j++) {
      temp += A.data[i * A.n_col + j] * X.data[j];
    }
    Y.data[i] = temp;
  }
}

__global__ void kernel_coalesce(const Matrix A_trans, const Vector X, Vector Y) {
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid > A_trans.n_col) {
    return;
  }
  for (int i = tid; i < A_trans.n_col; i += threadsPerBlock * blocksPerGrid) {
    float temp = 0;
    for (int j = 0; j < A_trans.n_row; j++) {
      temp += A_trans.data[i + j * A_trans.n_col] * X.data[j];
    }
    Y.data[i] = temp;
  }
}

__constant__ float DATA_CONSTANT[16384];
__global__ void kernel_constant(const Matrix A_trans, Vector Y) {
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid > A_trans.n_col) {
    return;
  }
  for (int i = tid; i < A_trans.n_col; i += threadsPerBlock * blocksPerGrid) {
    float temp = 0;
    for (int j = 0; j < A_trans.n_row; j++) {
      temp += A_trans.data[i + j * A_trans.n_col] * DATA_CONSTANT[j];
    }
    Y.data[i] = temp;
  }
}

__global__ void kernel_constant_loop_unroll(const Matrix A_trans, Vector Y) {
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid > A_trans.n_col) {
    return;
  }
  for (int i = tid; i < A_trans.n_col; i += threadsPerBlock * blocksPerGrid) {
    float temp = 0;
    for (int j = 0; j < A_trans.n_row; j += threadsPerBlock) {
      // loop unroll
      for (int k = 0; k < threadsPerBlock; ++k) {
        temp += A_trans.data[i + (j + k) * A_trans.n_col] * DATA_CONSTANT[j + k];
      }
    }
    for (int j = A_trans.n_row - A_trans.n_row % threadsPerBlock; j < A_trans.n_row; ++j) {
      temp += A_trans.data[i + j * A_trans.n_col] * DATA_CONSTANT[j];
    }
    Y.data[i] = temp;
  }
}

__global__ void kernel_shared(const Matrix A_trans, const Vector X, Vector Y) {
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int cid = threadIdx.x;
  __shared__ float cache[threadsPerBlock];
  const int cache_last = A_trans.n_row - A_trans.n_row % threadsPerBlock;
  // i: current col
  for (int i = tid; i < A_trans.n_col; i += threadsPerBlock * blocksPerGrid) {
    float temp = 0;
    // j: current row for load cache
    for (int j = cid; j < cache_last; j += threadsPerBlock) {
      __syncthreads();
      cache[cid] = X.data[j];
      __syncthreads();
      int begin = j - j % threadsPerBlock;
      // k: current row for calculate
      for (int k = 0; k < blockDim.x; ++k) {
        temp += A_trans.data[(k + begin) * A_trans.n_col + i] * cache[k];
      }
    }
    __syncthreads();
    if (cache_last + cid < A_trans.n_row) {
      cache[cid] = X.data[cache_last + cid];
    }
    __syncthreads();
    for (int k = cache_last; k < A_trans.n_row; k++) {
      temp += A_trans.data[k * A_trans.n_col + i] * cache[k - cache_last];
    }
    Y.data[i] = temp;
  }
}

__global__ void kernel_shared_loop_unroll(const Matrix A_trans, const Vector X, Vector Y) {
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int cid = threadIdx.x;
  __shared__ float cache[threadsPerBlock];
  const int cache_last = A_trans.n_row - A_trans.n_row % threadsPerBlock;
  // i: current col
  for (int i = tid; i < A_trans.n_col; i += threadsPerBlock * blocksPerGrid) {
    float temp = 0;
    // j: current row for load cache
    for (int j = cid; j < cache_last; j += threadsPerBlock) {
      __syncthreads();
      cache[cid] = X.data[j];
      __syncthreads();
      int begin = j - j % threadsPerBlock;
      // k: current row for calculate
      for (int k = 0; k < threadsPerBlock; ++k) {
        temp += A_trans.data[(k + begin) * A_trans.n_col + i] * cache[k];
      }
    }
    __syncthreads();
    if (cache_last + cid < A_trans.n_row) {
      cache[cid] = X.data[cache_last + cid];
    }
    __syncthreads();
    for (int k = cache_last; k < A_trans.n_row; k++) {
      temp += A_trans.data[k * A_trans.n_col + i] * cache[k - cache_last];
    }
    Y.data[i] = temp;
  }
}

__global__ void kernel_shared_loop_unroll_prefetch(const Matrix A_trans, const Vector X, Vector Y) {
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int cid = threadIdx.x;
  __shared__ float cache[threadsPerBlock];
  const int cache_last = A_trans.n_row - A_trans.n_row % threadsPerBlock;
  // i: current col
  for (int i = tid; i < A_trans.n_col; i += threadsPerBlock * blocksPerGrid) {
    float temp = 0;

    // prefetch
    __syncthreads();
    cache[cid] = X.data[cid];
    __syncthreads();
    // j: current row for load cache
    for (int j = cid + threadsPerBlock; j < cache_last; j += threadsPerBlock) {
      // prefetch
      float reg = X.data[j];
      int begin = j - threadsPerBlock - j % threadsPerBlock;
      // k: current row for calculate
      for (int k = 0; k < threadsPerBlock; ++k) {
        temp += A_trans.data[(k + begin) * A_trans.n_col + i] * cache[k];
      }
      __syncthreads();
      cache[cid] = reg;
      __syncthreads();
    }
    int begin = cache_last - threadsPerBlock;
    // k: current row for calculate
    for (int k = 0; k < threadsPerBlock; ++k) {
      temp += A_trans.data[(k + begin) * A_trans.n_col + i] * cache[k];
    }

    __syncthreads();
    if (cache_last + cid < A_trans.n_row) {
      cache[cid] = X.data[cache_last + cid];
    }
    __syncthreads();
    for (int k = cache_last; k < A_trans.n_row; k++) {
      temp += A_trans.data[k * A_trans.n_col + i] * cache[k - cache_last];
    }
    Y.data[i] = temp;
  }
}

__global__ void kernel_shuffle_loop_unroll(const Matrix A_trans, const Vector X, Vector Y) {
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int laneId = threadIdx.x % 32;
  const int cache_last = A_trans.n_row - A_trans.n_row % 32;
  // i: current col
  for (int i = tid; i < A_trans.n_col; i += threadsPerBlock * blocksPerGrid) {
    float temp = 0;
    // j: current row for load cache
    float shuffle_val = 0;
    for (int j = laneId; j < cache_last; j += 32) {
      shuffle_val = X.data[j];
      int cache_offset = j / 32 * 32;
      // k: current row for calculate
      for (int k = 0; k < 32; k++) {
        temp += A_trans.data[(cache_offset + k) * A_trans.n_col + i]
          * __shfl_sync(0xffffffff, shuffle_val, k, 32);
      }
    }
    if (cache_last + laneId < A_trans.n_row) {
      shuffle_val = X.data[cache_last + laneId];
    }
    for (int k = cache_last; k < A_trans.n_row; k++) {
      temp += A_trans.data[k * A_trans.n_col + i] 
        * __shfl_sync(0xffffffff, shuffle_val, k - cache_last, 32);
    }
    Y.data[i] = temp;
  }
}


void gemv(const Matrix A, const Vector X, Vector Y) {
  Matrix A_trans = transpose(A);
  Matrix d_A;
  Matrix d_A_trans;
  Vector d_X;
  Vector d_Y;
  d_A.n_col = A.n_col;
  d_A.n_row = A.n_row;
  d_A_trans.n_col = A_trans.n_col;
  d_A_trans.n_row = A_trans.n_row;
  d_X.length = X.length;
  d_Y.length = Y.length;
  int size_A = d_A.n_col * d_A.n_row;
  int size_X = d_X.length;
  int size_Y = d_Y.length;
  CUDA_CHECK(hipMalloc(&d_A.data, size_A * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_A_trans.data, size_A * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_X.data, size_X * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_Y.data, size_Y * sizeof(float)));
  CUDA_CHECK(hipMemcpy(d_A.data, A.data, size_A * sizeof(float),  hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_A_trans.data, A_trans.data, size_A*sizeof(float), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(DATA_CONSTANT), X.data, size_X*sizeof(float)));
  CUDA_CHECK(hipMemcpy(d_X.data, X.data, size_X * sizeof(float),  hipMemcpyHostToDevice));
  // invoke kernel
  dim3 dims_block(threadsPerBlock);
  dim3 dims_grid(blocksPerGrid);

  const int n_rounds = 10;

  // warm up
  kernel_naive<<<dims_grid, dims_block>>>(d_A, d_X, d_Y);

  // naive
  float elapsedTime_naive;
  hipEvent_t start_naive, stop_naive; 
  CUDA_CHECK(hipEventCreate(&start_naive)); 
  CUDA_CHECK(hipEventCreate(&stop_naive));
  CUDA_CHECK(hipEventRecord(start_naive, 0));
  for (int i = 0; i < n_rounds; ++i)
  kernel_naive<<<dims_grid, dims_block>>>(d_A, d_X, d_Y);
  CUDA_CHECK(hipEventRecord(stop_naive, 0)); 
  CUDA_CHECK(hipEventSynchronize(stop_naive)); 
  CUDA_CHECK(hipEventElapsedTime(&elapsedTime_naive, start_naive, stop_naive));
  CUDA_CHECK(hipEventDestroy(start_naive)); 
  CUDA_CHECK(hipEventDestroy(stop_naive));
  printf("Time of naive: %fms\n", elapsedTime_naive / n_rounds);
  // coalesce
  float elapsedTime_coalesce;
  hipEvent_t start_coalesce, stop_coalesce; 
  CUDA_CHECK(hipEventCreate(&start_coalesce)); 
  CUDA_CHECK(hipEventCreate(&stop_coalesce));
  CUDA_CHECK(hipEventRecord(start_coalesce, 0));
  for (int i = 0; i < n_rounds; ++i)
  kernel_coalesce<<<dims_grid, dims_block>>>(d_A_trans, d_X, d_Y);
  CUDA_CHECK(hipEventRecord(stop_coalesce, 0)); 
  CUDA_CHECK(hipEventSynchronize(stop_coalesce)); 
  CUDA_CHECK(hipEventElapsedTime(&elapsedTime_coalesce, start_coalesce, stop_coalesce));
  CUDA_CHECK(hipEventDestroy(start_coalesce)); 
  CUDA_CHECK(hipEventDestroy(stop_coalesce));
  printf("Time of coalesce: %fms\n", elapsedTime_coalesce / n_rounds);
  // constant
  float elapsedTime_constant;
  hipEvent_t start_constant, stop_constant; 
  CUDA_CHECK(hipEventCreate(&start_constant)); 
  CUDA_CHECK(hipEventCreate(&stop_constant));
  CUDA_CHECK(hipEventRecord(start_constant, 0));
  for (int i = 0; i < n_rounds; ++i)
  kernel_constant<<<dims_grid, dims_block>>>(d_A_trans, d_Y);
  CUDA_CHECK(hipEventRecord(stop_constant, 0)); 
  CUDA_CHECK(hipEventSynchronize(stop_constant)); 
  CUDA_CHECK(hipEventElapsedTime(&elapsedTime_constant, start_constant, stop_constant));
  CUDA_CHECK(hipEventDestroy(start_constant)); 
  CUDA_CHECK(hipEventDestroy(stop_constant));
  printf("Time of constant: %fms\n", elapsedTime_constant / n_rounds);
  // constant_loop_unroll
  float elapsedTime_constant_loop_unroll;
  hipEvent_t start_constant_loop_unroll, stop_constant_loop_unroll; 
  CUDA_CHECK(hipEventCreate(&start_constant_loop_unroll)); 
  CUDA_CHECK(hipEventCreate(&stop_constant_loop_unroll));
  CUDA_CHECK(hipEventRecord(start_constant_loop_unroll, 0));
  for (int i = 0; i < n_rounds; ++i)
  kernel_constant_loop_unroll<<<dims_grid, dims_block>>>(d_A_trans, d_Y);
  CUDA_CHECK(hipEventRecord(stop_constant_loop_unroll, 0)); 
  CUDA_CHECK(hipEventSynchronize(stop_constant_loop_unroll)); 
  CUDA_CHECK(hipEventElapsedTime(&elapsedTime_constant_loop_unroll, start_constant_loop_unroll, stop_constant_loop_unroll));
  CUDA_CHECK(hipEventDestroy(start_constant_loop_unroll)); 
  CUDA_CHECK(hipEventDestroy(stop_constant_loop_unroll));
  printf("Time of constant_loop_unroll: %fms\n", elapsedTime_constant_loop_unroll / n_rounds);
  // shared
  float elapsedTime_shared;
  hipEvent_t start_shared, stop_shared; 
  CUDA_CHECK(hipEventCreate(&start_shared)); 
  CUDA_CHECK(hipEventCreate(&stop_shared));
  CUDA_CHECK(hipEventRecord(start_shared, 0));
  for (int i = 0; i < n_rounds; ++i)
  kernel_shared<<<dims_grid, dims_block>>>(d_A_trans, d_X, d_Y);
  CUDA_CHECK(hipEventRecord(stop_shared, 0)); 
  CUDA_CHECK(hipEventSynchronize(stop_shared)); 
  CUDA_CHECK(hipEventElapsedTime(&elapsedTime_shared, start_shared, stop_shared));
  CUDA_CHECK(hipEventDestroy(start_shared)); 
  CUDA_CHECK(hipEventDestroy(stop_shared));
  printf("Time of shared: %fms\n", elapsedTime_shared / n_rounds);
  // shared_loop_unroll
  float elapsedTime_shared_loop_unroll;
  hipEvent_t start_shared_loop_unroll, stop_shared_loop_unroll; 
  CUDA_CHECK(hipEventCreate(&start_shared_loop_unroll)); 
  CUDA_CHECK(hipEventCreate(&stop_shared_loop_unroll));
  CUDA_CHECK(hipEventRecord(start_shared_loop_unroll, 0));
  for (int i = 0; i < n_rounds; ++i)
  kernel_shared_loop_unroll<<<dims_grid, dims_block>>>(d_A_trans, d_X, d_Y);
  CUDA_CHECK(hipEventRecord(stop_shared_loop_unroll, 0)); 
  CUDA_CHECK(hipEventSynchronize(stop_shared_loop_unroll)); 
  CUDA_CHECK(hipEventElapsedTime(&elapsedTime_shared_loop_unroll, start_shared_loop_unroll, stop_shared_loop_unroll));
  CUDA_CHECK(hipEventDestroy(start_shared_loop_unroll)); 
  CUDA_CHECK(hipEventDestroy(stop_shared_loop_unroll));
  printf("Time of shared_loop_unroll: %fms\n", elapsedTime_shared_loop_unroll / n_rounds);
  // shared_loop_unroll_prefetch
  float elapsedTime_shared_loop_unroll_prefetch;
  hipEvent_t start_shared_loop_unroll_prefetch, stop_shared_loop_unroll_prefetch; 
  CUDA_CHECK(hipEventCreate(&start_shared_loop_unroll_prefetch)); 
  CUDA_CHECK(hipEventCreate(&stop_shared_loop_unroll_prefetch));
  CUDA_CHECK(hipEventRecord(start_shared_loop_unroll_prefetch, 0));
  for (int i = 0; i < n_rounds; ++i)
  kernel_shared_loop_unroll_prefetch<<<dims_grid, dims_block>>>(d_A_trans, d_X, d_Y);
  CUDA_CHECK(hipEventRecord(stop_shared_loop_unroll_prefetch, 0)); 
  CUDA_CHECK(hipEventSynchronize(stop_shared_loop_unroll_prefetch)); 
  CUDA_CHECK(hipEventElapsedTime(&elapsedTime_shared_loop_unroll_prefetch, start_shared_loop_unroll_prefetch, stop_shared_loop_unroll_prefetch));
  CUDA_CHECK(hipEventDestroy(start_shared_loop_unroll_prefetch)); 
  CUDA_CHECK(hipEventDestroy(stop_shared_loop_unroll_prefetch));
  printf("Time of shared_loop_unroll_prefetch: %fms\n", elapsedTime_shared_loop_unroll_prefetch / n_rounds);
  // shuffle_loop_unroll
  float elapsedTime_shuffle_loop_unroll;
  hipEvent_t start_shuffle_loop_unroll, stop_shuffle_loop_unroll; 
  CUDA_CHECK(hipEventCreate(&start_shuffle_loop_unroll)); 
  CUDA_CHECK(hipEventCreate(&stop_shuffle_loop_unroll));
  CUDA_CHECK(hipEventRecord(start_shuffle_loop_unroll, 0));
  for (int i = 0; i < n_rounds; ++i)
  kernel_shuffle_loop_unroll<<<dims_grid, dims_block>>>(d_A_trans, d_X, d_Y);
  CUDA_CHECK(hipEventRecord(stop_shuffle_loop_unroll, 0)); 
  CUDA_CHECK(hipEventSynchronize(stop_shuffle_loop_unroll)); 
  CUDA_CHECK(hipEventElapsedTime(&elapsedTime_shuffle_loop_unroll, start_shuffle_loop_unroll, stop_shuffle_loop_unroll));
  CUDA_CHECK(hipEventDestroy(start_shuffle_loop_unroll)); 
  CUDA_CHECK(hipEventDestroy(stop_shuffle_loop_unroll));
  printf("Time of shuffle_loop_unroll: %fms\n", elapsedTime_shuffle_loop_unroll / n_rounds);
  // cublas
  float elapsedTime_cublas;
  hipEvent_t start_cublas, stop_cublas; 
  CUDA_CHECK(hipEventCreate(&start_cublas)); 
  CUDA_CHECK(hipEventCreate(&stop_cublas));
  hipblasHandle_t handle;
  (hipblasCreate(&handle));
  (hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST));
  // transpose by default because of column priority
  hipblasOperation_t trans = HIPBLAS_OP_N;
  const int dim0_tensor1 = d_A_trans.n_row;
  const int dim1_tensor1 = d_A_trans.n_col;
  const int lda = dim1_tensor1;
  const int incx = 1;
  const int incy = 1;
  const float *a = d_A_trans.data;
  const float *x = d_X.data;
  float *y = d_Y.data;
  float alpha = 1, beta = 0;
  CUDA_CHECK(hipEventRecord(start_cublas, 0));
  for (int i = 0; i < n_rounds; ++i)
  (hipblasSgemv(handle, trans, dim1_tensor1, dim0_tensor1, 
              &alpha, a, lda, x, incx, &beta, y, incy));
  CUDA_CHECK(hipEventRecord(stop_cublas, 0)); 
  CUDA_CHECK(hipEventSynchronize(stop_cublas)); 
  CUDA_CHECK(hipEventElapsedTime(&elapsedTime_cublas, start_cublas, stop_cublas));
  CUDA_CHECK(hipEventDestroy(start_cublas)); 
  CUDA_CHECK(hipEventDestroy(stop_cublas));
  printf("Time of cublas: %fms\n", elapsedTime_cublas / n_rounds);
  
  // copy data from device to host
  CUDA_CHECK(hipMemcpy(Y.data, d_Y.data, size_Y * sizeof(float),  hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(d_A.data));
  CUDA_CHECK(hipFree(d_X.data));
  CUDA_CHECK(hipFree(d_Y.data));
  free(A_trans.data);
  return;
}