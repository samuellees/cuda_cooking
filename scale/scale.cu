#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "assert.h"
#include <cmath>

#include <hipblas.h>
#include "hip/hip_runtime.h"

const char* cublasGetErrorString(hipblasStatus_t error) {
  switch (error) {
  case HIPBLAS_STATUS_SUCCESS:
    return "CUBLAS_STATUS_SUCCESS";
  case HIPBLAS_STATUS_NOT_INITIALIZED:
    return "CUBLAS_STATUS_NOT_INITIALIZED";
  case HIPBLAS_STATUS_ALLOC_FAILED:
    return "CUBLAS_STATUS_ALLOC_FAILED";
  case HIPBLAS_STATUS_INVALID_VALUE:
    return "CUBLAS_STATUS_INVALID_VALUE";
  case HIPBLAS_STATUS_ARCH_MISMATCH:
    return "CUBLAS_STATUS_ARCH_MISMATCH";
  case HIPBLAS_STATUS_INTERNAL_ERROR:
    return "CUBLAS_STATUS_INTERNAL_ERROR";
  case HIPBLAS_STATUS_NOT_SUPPORTED:
    return "CUBLAS_STATUS_NOT_SUPPORTED";
  case HIPBLAS_STATUS_UNKNOWN:
    return "CUBLAS_STATUS_LICENSE_ERROR";
  }
  // To suppress compiler warning.
  return "Unrecognized cublas error string";
}

#define CUDA_CHECK(code)                                                  \
  {                                                                       \
    hipError_t status = (code);                                         \
    if ((status) != hipSuccess) {                                          \
      fprintf(stderr, "CUDA error in file: %s, line: %d, %s\n", __FILE__, \
              __LINE__, hipGetErrorString((status)));                      \
      exit((status));                                                       \
    }                                                                     \
  }

#define CUBLAS_CHECK(code)                                           \
  {                                                                       \
    hipblasStatus_t status = (code);                                         \
    if ((status) != HIPBLAS_STATUS_SUCCESS) {                                          \
      fprintf(stderr, "cublas error in file: %s, line: %d, %s\n", __FILE__, \
              __LINE__, cublasGetErrorString((status)));                      \
      exit((status));                                                       \
    }                                                                     \
  }

template<typename scalar_t>
void print_data(const scalar_t * data, int length) {
  for (int i = 0; i < length; ++i) {
      std::cout << data[i] << ", ";
  }
  std::cout << std::endl;
}

int main() {
  int size = 6;
  float *input = new float[size];
  float alpha = 2.0;
  float *input_dev = nullptr;
  float *alpha_dev = nullptr;

  for (int i = 0; i < size; ++i) input[i] = 2.1;
  
  CUDA_CHECK(hipMalloc(&input_dev, sizeof(float) * size));
  CUDA_CHECK(hipMemcpy(input_dev, input, sizeof(float) * size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMalloc(&alpha_dev, sizeof(float) * 1));
  CUDA_CHECK(hipMemcpy(alpha_dev, &alpha, sizeof(float) * 1, hipMemcpyHostToDevice));

  hipblasHandle_t handle;
  CUBLAS_CHECK(hipblasCreate(&handle));
  CUBLAS_CHECK(hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST));
  CUBLAS_CHECK(hipblasScalEx(handle, size, &alpha, HIP_R_32F, input_dev, HIP_R_32F, 1, HIP_R_32F));
  // CUBLAS_CHECK(cublasSetPointerMode(handle, CUBLAS_POINTER_MODE_DEVICE));
  // CUBLAS_CHECK(cublasScalEx(handle, size, alpha_dev, CUDA_R_32F, input_dev, CUDA_R_32F, 1, CUDA_R_32F));
  
  
  CUDA_CHECK(hipMemcpy(input, input_dev, sizeof(float) * size, hipMemcpyDeviceToHost));

  print_data(input, size);

  
  CUDA_CHECK(hipFree(input_dev));
  CUDA_CHECK(hipFree(alpha_dev));
  delete []input;

  return 0;
}