#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "gemm.h"
#include "gemm_kernels.cuh"
#include "coding.cuh"
#include "utils.cuh"
#include <hipblas.h>

#define WARMUP 1
// #define K1 1
// #define K2 1
// #define K3 1
// #define K4 1
// #define K5 1
// #define K6 1
#define K7 1
// #define K8 1

double getGFlops(double time_ms, int64_t m, int64_t n, int64_t k) {
  return 2 * m * k * n / (time_ms/1000) *1e-9;
}

const int n_rounds = 10;

void gemm(const Matrix A, const Matrix B, Matrix C, std::vector<float>& flops_info) {
  // copy A, B and C to device
  Matrix d_A = A;
  Matrix d_B = B;
  Matrix d_C = C;
  int64_t size_A = d_A.n_col * d_A.n_row;
  int64_t size_B = d_B.n_col * d_B.n_row;
  int64_t size_C = d_C.n_col * d_C.n_row;
  CUDA_CHECK(hipMalloc(&d_A.data, size_A * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_B.data, size_B * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_C.data, size_C * sizeof(float)));
  CUDA_CHECK(hipMemcpy(d_A.data, A.data, size_A * sizeof(float),  hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_B.data, B.data, size_B * sizeof(float),  hipMemcpyHostToDevice));
  
  // invoke kernel
  dim3 dims_block(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dims_grid(CEIL_DIV(C.n_col, dims_block.x), CEIL_DIV(C.n_row, dims_block.y));

#ifdef WARMUP
  // warm up
  kernel_naive<<<dims_grid, dims_block>>>(
      d_A.n_row, d_B.n_col, d_A.n_col, d_A.data, d_B.data, d_C.data);
#endif

#ifdef K1
  // 1 naive
  float elapsedTime_naive;
  hipEvent_t start_naive, stop_naive; 
  CUDA_CHECK(hipEventCreate(&start_naive)); 
  CUDA_CHECK(hipEventCreate(&stop_naive));
  CUDA_CHECK(hipEventRecord(start_naive, 0));
  for (int i = 0; i < n_rounds; ++i) {
    kernel_naive<<<dims_grid, dims_block>>>(
        d_A.n_row, d_B.n_col, d_A.n_col, d_A.data, d_B.data, d_C.data);
  }
  CUDA_CHECK(hipEventRecord(stop_naive, 0)); 
  CUDA_CHECK(hipEventSynchronize(stop_naive)); 
  CUDA_CHECK(hipEventElapsedTime(&elapsedTime_naive, start_naive, stop_naive));
  CUDA_CHECK(hipEventDestroy(start_naive)); 
  CUDA_CHECK(hipEventDestroy(stop_naive));
  elapsedTime_naive = elapsedTime_naive / n_rounds;
  float flops_naive = getGFlops(elapsedTime_naive, A.n_row, B.n_col, A.n_col);
  printf("kernel %-20s: %8.2f ms, %8.2f GFlops, %6.2f%% of cublas.\n", 
            "naive", 
            elapsedTime_naive, flops_naive, 
            flops_naive / flops_info[0] * 100);
  flops_info.push_back(flops_naive);
#endif


#ifdef K2
  // 2 shared
  float elapsedTime_shared;
  hipEvent_t start_shared, stop_shared; 
  CUDA_CHECK(hipEventCreate(&start_shared)); 
  CUDA_CHECK(hipEventCreate(&stop_shared));
  CUDA_CHECK(hipEventRecord(start_shared, 0));
  for (int i = 0; i < n_rounds; ++i) {
    kernel_shared<<<dims_grid, dims_block>>>(
        d_A.n_row, d_B.n_col, d_A.n_col, d_A.data, d_B.data, d_C.data);
  }
  CUDA_CHECK(hipEventRecord(stop_shared, 0)); 
  CUDA_CHECK(hipEventSynchronize(stop_shared)); 
  CUDA_CHECK(hipEventElapsedTime(&elapsedTime_shared, start_shared, stop_shared));
  CUDA_CHECK(hipEventDestroy(start_shared)); 
  CUDA_CHECK(hipEventDestroy(stop_shared));
  elapsedTime_shared = elapsedTime_shared / n_rounds;
  float flops_shared = getGFlops(elapsedTime_shared, A.n_row, B.n_col, A.n_col);
  printf("kernel %-20s: %8.2f ms, %8.2f GFlops, %6.2f%% of cublas.\n", 
            "shared", 
            elapsedTime_shared, flops_shared, 
            flops_shared / flops_info[0] * 100);
  flops_info.push_back(flops_shared);
#endif


#ifdef K3
  // 3 shared_4workloads
  float elapsedTime_shared_4w;
  dim3 dims_block_shared_4w(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dims_grid_shared_4w(CEIL_DIV(C.n_col, BLOCK_SIZE*2), CEIL_DIV(C.n_row, BLOCK_SIZE*2));
  hipEvent_t start_shared_4w, stop_shared_4w; 
  CUDA_CHECK(hipEventCreate(&start_shared_4w)); 
  CUDA_CHECK(hipEventCreate(&stop_shared_4w));
  CUDA_CHECK(hipEventRecord(start_shared_4w, 0));
  for (int i = 0; i < n_rounds; ++i) {
    kernel_shared_4w<<<dims_grid_shared_4w, dims_block_shared_4w>>>(
        d_A.n_row, d_B.n_col, d_A.n_col, d_A.data, d_B.data, d_C.data);
  }
  CUDA_CHECK(hipEventRecord(stop_shared_4w, 0)); 
  CUDA_CHECK(hipEventSynchronize(stop_shared_4w)); 
  CUDA_CHECK(hipEventElapsedTime(&elapsedTime_shared_4w, start_shared_4w, stop_shared_4w));
  CUDA_CHECK(hipEventDestroy(start_shared_4w)); 
  CUDA_CHECK(hipEventDestroy(stop_shared_4w));
  elapsedTime_shared_4w = elapsedTime_shared_4w / n_rounds;
  float flops_shared_4w = getGFlops(elapsedTime_shared_4w, A.n_row, B.n_col, A.n_col);
  printf("kernel %-20s: %8.2f ms, %8.2f GFlops, %6.2f%% of cublas.\n", 
          "shared_4w", 
          elapsedTime_shared_4w, flops_shared_4w, 
          flops_shared_4w / flops_info[0] * 100);
  flops_info.push_back(flops_shared_4w);
#endif


  // padding vars
  float time_padding = 0;
  float time_unpadding = 0;
  float * d_padA = nullptr;
  float * d_padB = nullptr;
  float * d_padC = nullptr;
  int padM = 0;
  int padN = 0;
  int padK = 0;


#ifdef K4
  // 4 shared_4workloads_padding
  time_padding = 0;
  padM = CEIL_DIV(C.n_row, BLOCK_SIZE_L) * BLOCK_SIZE_L;
  padN = CEIL_DIV(C.n_col, BLOCK_SIZE_L) * BLOCK_SIZE_L;
  padK = CEIL_DIV(B.n_row, BLOCK_SIZE_L) * BLOCK_SIZE_L;
  CUDA_CHECK(hipMalloc(&d_padA, padM * padK * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_padB, padK * padN * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_padC, padM * padN * sizeof(float)));
  time_padding += padding(d_A.data, d_padA, A.n_row, A.n_col, padM, padK);
  time_padding += padding(d_B.data, d_padB, B.n_row, B.n_col, padK, padN);

  float elapsedTime_shared_4w_pad;
  dim3 dims_block_shared_4w_pad(BLOCK_SIZE_L/2, BLOCK_SIZE_L/2);
  dim3 dims_grid_shared_4w_pad(CEIL_DIV(C.n_col, BLOCK_SIZE_L), CEIL_DIV(C.n_row, BLOCK_SIZE_L));
  hipEvent_t start_shared_4w_pad, stop_shared_4w_pad; 
  CUDA_CHECK(hipEventCreate(&start_shared_4w_pad)); 
  CUDA_CHECK(hipEventCreate(&stop_shared_4w_pad));
  CUDA_CHECK(hipEventRecord(start_shared_4w_pad, 0));
  for (int i = 0; i < n_rounds; ++i) {
    kernel_shared_4w_pad<<<dims_grid_shared_4w_pad, dims_block_shared_4w_pad>>>(
      padM, padN, padK, d_padA, d_padB, d_padC);
  }
  CUDA_CHECK(hipEventRecord(stop_shared_4w_pad, 0)); 
  CUDA_CHECK(hipEventSynchronize(stop_shared_4w_pad)); 
  CUDA_CHECK(hipEventElapsedTime(&elapsedTime_shared_4w_pad, start_shared_4w_pad, stop_shared_4w_pad));
  CUDA_CHECK(hipEventDestroy(start_shared_4w_pad)); 
  CUDA_CHECK(hipEventDestroy(stop_shared_4w_pad));
  time_unpadding = unpadding(d_C.data, d_padC, d_C.n_row, d_C.n_col, padM, padN);
  CUDA_CHECK(hipFree(d_padA));
  CUDA_CHECK(hipFree(d_padB));
  CUDA_CHECK(hipFree(d_padC));
  elapsedTime_shared_4w_pad = elapsedTime_shared_4w_pad / n_rounds + time_padding+time_unpadding;
  float flops_shared_4w_pad = getGFlops(elapsedTime_shared_4w_pad, A.n_row, B.n_col, A.n_col);
  printf("kernel %-20s: %8.2f ms, %8.2f GFlops, %6.2f%% of cublas.\n", 
        "shared_4w_pad", 
        elapsedTime_shared_4w_pad, flops_shared_4w_pad, 
        flops_shared_4w_pad / flops_info[0] * 100);
  flops_info.push_back(flops_shared_4w_pad);
#endif


#ifdef K5
  // 5 shared_8workloads_padding
  time_padding = 0;
  padM = CEIL_DIV(C.n_row, BLOCK_SIZE) * BLOCK_SIZE;
  padN = CEIL_DIV(C.n_col, BLOCK_SIZE) * BLOCK_SIZE;
  padK = CEIL_DIV(B.n_row, BLOCK_SIZE) * BLOCK_SIZE;
  CUDA_CHECK(hipMalloc(&d_padA, padM * padK * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_padB, padK * padN * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_padC, padM * padN * sizeof(float)));
  time_padding += padding(d_A.data, d_padA, A.n_row, A.n_col, padM, padK);
  time_padding += padding(d_B.data, d_padB, B.n_row, B.n_col, padK, padN);

  float elapsedTime_shared_8w_pad;
  dim3 dims_block_shared_8w_pad(BLOCK_SIZE, BLOCK_SIZE/WORK_PERTHREAD);
  dim3 dims_grid_shared_8w_pad(CEIL_DIV(C.n_col, BLOCK_SIZE), CEIL_DIV(C.n_row, BLOCK_SIZE));
  hipEvent_t start_shared_8w_pad, stop_shared_8w_pad; 
  CUDA_CHECK(hipEventCreate(&start_shared_8w_pad)); 
  CUDA_CHECK(hipEventCreate(&stop_shared_8w_pad));
  CUDA_CHECK(hipEventRecord(start_shared_8w_pad, 0));
  for (int i = 0; i < n_rounds; ++i) {
    kernel_shared_8w_pad<<<dims_grid_shared_8w_pad, dims_block_shared_8w_pad>>>(
      padM, padN, padK, d_padA, d_padB, d_padC);
  }
  CUDA_CHECK(hipEventRecord(stop_shared_8w_pad, 0)); 
  CUDA_CHECK(hipEventSynchronize(stop_shared_8w_pad)); 
  CUDA_CHECK(hipEventElapsedTime(&elapsedTime_shared_8w_pad, start_shared_8w_pad, stop_shared_8w_pad));
  CUDA_CHECK(hipEventDestroy(start_shared_8w_pad)); 
  CUDA_CHECK(hipEventDestroy(stop_shared_8w_pad));
  time_unpadding = unpadding(d_C.data, d_padC, d_C.n_row, d_C.n_col, padM, padN);
  CUDA_CHECK(hipFree(d_padA));
  CUDA_CHECK(hipFree(d_padB));
  CUDA_CHECK(hipFree(d_padC));
  elapsedTime_shared_8w_pad = elapsedTime_shared_8w_pad / n_rounds + time_padding+time_unpadding;
  float flops_shared_8w_pad = getGFlops(elapsedTime_shared_8w_pad, A.n_row, B.n_col, A.n_col);
  printf("kernel %-20s: %8.2f ms, %8.2f GFlops, %6.2f%% of cublas.\n", 
        "shared_8w_pad", 
        elapsedTime_shared_8w_pad, flops_shared_8w_pad, 
        flops_shared_8w_pad / flops_info[0] * 100);
  flops_info.push_back(flops_shared_8w_pad);
#endif


#ifdef K6
  // 6 shared_32workloads2D_padding
  time_padding = 0;
  padM = CEIL_DIV(C.n_row, BLOCK_SIZE_L) * BLOCK_SIZE_L;
  padN = CEIL_DIV(C.n_col, BLOCK_SIZE_L) * BLOCK_SIZE_L;
  padK = CEIL_DIV(B.n_row, BLOCK_SIZE_L) * BLOCK_SIZE_L;
  CUDA_CHECK(hipMalloc(&d_padA, padM * padK * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_padB, padK * padN * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_padC, padM * padN * sizeof(float)));
  time_padding += padding(d_A.data, d_padA, A.n_row, A.n_col, padM, padK);
  time_padding += padding(d_B.data, d_padB, B.n_row, B.n_col, padK, padN);

  float elapsedTime_shared_32w2d_pad;
  dim3 dims_block_shared_32w2d_pad(NTX, NTY);
  dim3 dims_grid_shared_32w2d_pad(CEIL_DIV(C.n_col, BLOCK_SIZE_L), CEIL_DIV(C.n_row, BLOCK_SIZE_L));
  hipEvent_t start_shared_32w2d_pad, stop_shared_32w2d_pad; 
  CUDA_CHECK(hipEventCreate(&start_shared_32w2d_pad)); 
  CUDA_CHECK(hipEventCreate(&stop_shared_32w2d_pad));
  CUDA_CHECK(hipEventRecord(start_shared_32w2d_pad, 0));
  for (int i = 0; i < n_rounds; ++i) {
    kernel_shared_32w2d_pad<<<dims_grid_shared_32w2d_pad, dims_block_shared_32w2d_pad>>>(
      padM, padN, padK, d_padA, d_padB, d_padC);
  }
  CUDA_CHECK(hipEventRecord(stop_shared_32w2d_pad, 0)); 
  CUDA_CHECK(hipEventSynchronize(stop_shared_32w2d_pad)); 
  CUDA_CHECK(hipEventElapsedTime(&elapsedTime_shared_32w2d_pad, start_shared_32w2d_pad, stop_shared_32w2d_pad));
  CUDA_CHECK(hipEventDestroy(start_shared_32w2d_pad)); 
  CUDA_CHECK(hipEventDestroy(stop_shared_32w2d_pad));
  time_unpadding = unpadding(d_C.data, d_padC, d_C.n_row, d_C.n_col, padM, padN);
  CUDA_CHECK(hipFree(d_padA));
  CUDA_CHECK(hipFree(d_padB));
  CUDA_CHECK(hipFree(d_padC));
  elapsedTime_shared_32w2d_pad = elapsedTime_shared_32w2d_pad / n_rounds + time_padding+time_unpadding;
  float flops_shared_32w2d_pad = getGFlops(elapsedTime_shared_32w2d_pad, A.n_row, B.n_col, A.n_col);
  printf("kernel %-20s: %8.2f ms, %8.2f GFlops, %6.2f%% of cublas.\n", 
      "shared_32w2d_pad", 
      elapsedTime_shared_32w2d_pad, flops_shared_32w2d_pad, 
      flops_shared_32w2d_pad / flops_info[0] * 100);
  flops_info.push_back(flops_shared_32w2d_pad);
#endif


#ifdef K7
  // 7 shared_32workloads2D_padding_vec
  time_padding = 0;
  padM = CEIL_DIV(C.n_row, BLOCK_SIZE_L) * BLOCK_SIZE_L;
  padN = CEIL_DIV(C.n_col, BLOCK_SIZE_L) * BLOCK_SIZE_L;
  padK = CEIL_DIV(B.n_row, BLOCK_SIZE_L) * BLOCK_SIZE_L;
  CUDA_CHECK(hipMalloc(&d_padA, padM * padK * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_padB, padK * padN * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_padC, padM * padN * sizeof(float)));
  time_padding += padding(d_A.data, d_padA, A.n_row, A.n_col, padM, padK);
  time_padding += padding(d_B.data, d_padB, B.n_row, B.n_col, padK, padN);
  float elapsedTime_shared_32w2d_pad_vec;
  dim3 dims_block_shared_32w2d_pad_vec(NTX, NTY);
  dim3 dims_grid_shared_32w2d_pad_vec(CEIL_DIV(C.n_col, BLOCK_SIZE_L), CEIL_DIV(C.n_row, BLOCK_SIZE_L));
  hipEvent_t start_shared_32w2d_pad_vec, stop_shared_32w2d_pad_vec; 
    kernel_shared_32w2d_pad_vec<<<dims_grid_shared_32w2d_pad_vec, dims_block_shared_32w2d_pad_vec>>>(
      padM, padN, padK, (float4*)d_padA, (float4*)d_padB, d_padC);
  CUDA_CHECK(hipEventCreate(&start_shared_32w2d_pad_vec)); 
  CUDA_CHECK(hipEventCreate(&stop_shared_32w2d_pad_vec));
  CUDA_CHECK(hipEventRecord(start_shared_32w2d_pad_vec, 0));
  for (int i = 0; i < n_rounds; ++i) {
    kernel_shared_32w2d_pad_vec<<<dims_grid_shared_32w2d_pad_vec, dims_block_shared_32w2d_pad_vec>>>(
      padM, padN, padK, (float4*)d_padA, (float4*)d_padB, d_padC);
  }
  CUDA_CHECK(hipEventRecord(stop_shared_32w2d_pad_vec, 0)); 
  CUDA_CHECK(hipEventSynchronize(stop_shared_32w2d_pad_vec)); 
  CUDA_CHECK(hipEventElapsedTime(&elapsedTime_shared_32w2d_pad_vec, start_shared_32w2d_pad_vec, stop_shared_32w2d_pad_vec));
  CUDA_CHECK(hipEventDestroy(start_shared_32w2d_pad_vec)); 
  CUDA_CHECK(hipEventDestroy(stop_shared_32w2d_pad_vec));
  time_unpadding = unpadding(d_C.data, d_padC, d_C.n_row, d_C.n_col, padM, padN);
  CUDA_CHECK(hipFree(d_padA));
  CUDA_CHECK(hipFree(d_padB));
  CUDA_CHECK(hipFree(d_padC));
  elapsedTime_shared_32w2d_pad_vec = elapsedTime_shared_32w2d_pad_vec / n_rounds + time_padding+time_unpadding;
  // elapsedTime_shared_32w2d_pad_vec = elapsedTime_shared_32w2d_pad_vec / n_rounds;
  float flops_shared_32w2d_pad_vec = getGFlops(elapsedTime_shared_32w2d_pad_vec, A.n_row, B.n_col, A.n_col);
  printf("kernel %-20s: %8.2f ms, %8.2f GFlops, %6.2f%% of cublas.\n", 
    "shared_32w2d_pad_vec", 
    elapsedTime_shared_32w2d_pad_vec, flops_shared_32w2d_pad_vec, 
    flops_shared_32w2d_pad_vec / flops_info[0] * 100);
  flops_info.push_back(flops_shared_32w2d_pad_vec);
#endif


#ifdef K8
  // 8 shared_64workloads2D_padding_vec
  time_padding = 0;
  padM = CEIL_DIV(C.n_row, BLOCK_SIZE_L_MAX) * BLOCK_SIZE_L_MAX;
  padN = CEIL_DIV(C.n_col, BLOCK_SIZE_L_MAX) * BLOCK_SIZE_L_MAX;
  padK = CEIL_DIV(B.n_row, BLOCK_SIZE_L_MAX) * BLOCK_SIZE_L_MAX;
  CUDA_CHECK(hipMalloc(&d_padA, padM * padK * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_padB, padK * padN * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_padC, padM * padN * sizeof(float)));
  time_padding += padding(d_A.data, d_padA, A.n_row, A.n_col, padM, padK);
  time_padding += padding(d_B.data, d_padB, B.n_row, B.n_col, padK, padN);
  float elapsedTime_shared_64w2d_pad_vec;
  dim3 dims_block_shared_64w2d_pad_vec(NTX_MAX, NTY_MAX);
  dim3 dims_grid_shared_64w2d_pad_vec(CEIL_DIV(C.n_col, BLOCK_SIZE_L_MAX), CEIL_DIV(C.n_row, BLOCK_SIZE_L_MAX));
  hipEvent_t start_shared_64w2d_pad_vec, stop_shared_64w2d_pad_vec; 
    kernel_shared_64w2d_pad_vec<<<dims_grid_shared_64w2d_pad_vec, dims_block_shared_64w2d_pad_vec>>>(
      padM, padN, padK, (float4*)d_padA, (float4*)d_padB, d_padC);
  CUDA_CHECK(hipEventCreate(&start_shared_64w2d_pad_vec)); 
  CUDA_CHECK(hipEventCreate(&stop_shared_64w2d_pad_vec));
  CUDA_CHECK(hipEventRecord(start_shared_64w2d_pad_vec, 0));
  for (int i = 0; i < n_rounds; ++i) {
    kernel_shared_64w2d_pad_vec<<<dims_grid_shared_64w2d_pad_vec, dims_block_shared_64w2d_pad_vec>>>(
      padM, padN, padK, (float4*)d_padA, (float4*)d_padB, d_padC);
  }
  CUDA_CHECK(hipEventRecord(stop_shared_64w2d_pad_vec, 0)); 
  CUDA_CHECK(hipEventSynchronize(stop_shared_64w2d_pad_vec)); 
  CUDA_CHECK(hipEventElapsedTime(&elapsedTime_shared_64w2d_pad_vec, start_shared_64w2d_pad_vec, stop_shared_64w2d_pad_vec));
  CUDA_CHECK(hipEventDestroy(start_shared_64w2d_pad_vec)); 
  CUDA_CHECK(hipEventDestroy(stop_shared_64w2d_pad_vec));
  time_unpadding = unpadding(d_C.data, d_padC, d_C.n_row, d_C.n_col, padM, padN);
  CUDA_CHECK(hipFree(d_padA));
  CUDA_CHECK(hipFree(d_padB));
  CUDA_CHECK(hipFree(d_padC));
  elapsedTime_shared_64w2d_pad_vec = elapsedTime_shared_64w2d_pad_vec / n_rounds + time_padding+time_unpadding;
  float flops_shared_64w2d_pad_vec = getGFlops(elapsedTime_shared_64w2d_pad_vec, A.n_row, B.n_col, A.n_col);
  printf("kernel %-20s: %8.2f ms, %8.2f GFlops, %6.2f%% of cublas.\n", 
    "shared_64w2d_pad_vec", 
    elapsedTime_shared_64w2d_pad_vec, flops_shared_64w2d_pad_vec, 
    flops_shared_64w2d_pad_vec / flops_info[0] * 100);
  flops_info.push_back(flops_shared_64w2d_pad_vec);
#endif


  // copy data from device to host
  CUDA_CHECK(hipMemcpy(C.data, d_C.data, size_C * sizeof(float),  hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(d_A.data));
  CUDA_CHECK(hipFree(d_B.data));
  CUDA_CHECK(hipFree(d_C.data));
}


void gemm_ref(const Matrix A, const Matrix B, Matrix C, std::vector<float>& flops_info) {
  // copy A, B and C to device
  Matrix d_A = A;
  Matrix d_B = B;
  Matrix d_C = C;
  int64_t size_A = d_A.n_col * d_A.n_row;
  int64_t size_B = d_B.n_col * d_B.n_row;
  int64_t size_C = d_C.n_col * d_C.n_row;
  CUDA_CHECK(hipMalloc(&d_A.data, size_A * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_B.data, size_B * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_C.data, size_C * sizeof(float)));
  CUDA_CHECK(hipMemcpy(d_A.data, A.data, size_A * sizeof(float),  hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_B.data, B.data, size_B * sizeof(float),  hipMemcpyHostToDevice));

  float elapsedTime_cublas;
  hipEvent_t start_cublas, stop_cublas; 
  CUDA_CHECK(hipEventCreate(&start_cublas)); 
  CUDA_CHECK(hipEventCreate(&stop_cublas));
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
  hipblasOperation_t transa = HIPBLAS_OP_N;
  hipblasOperation_t transb = HIPBLAS_OP_N;
  const int M = A.n_row;
  const int K = A.n_col;
  const int N = B.n_col;
  const int m = N;
  const int k = K;
  const int n = M;
  const int lda = A.n_col;
  const int ldb = B.n_col;
  const int ldc = B.n_col;
  using scalar_t = float;
  const scalar_t *a = d_A.data;
  const scalar_t *b = d_B.data;
  scalar_t *c = d_C.data;
  scalar_t alpha = 1, beta = 0;
  // warm up
  hipblasSgemm(handle, transb, transa, m, n, k,
                      &alpha, b, ldb, a, lda, &beta, c, ldc);
  CUDA_CHECK(hipEventRecord(start_cublas, 0));
  for (int i = 0; i < n_rounds; ++i)
    hipblasSgemm(handle, transb, transa, m, n, k,
                        &alpha, b, ldb, a, lda, &beta, c, ldc);
  CUDA_CHECK(hipEventRecord(stop_cublas, 0)); 
  CUDA_CHECK(hipEventSynchronize(stop_cublas)); 
  CUDA_CHECK(hipEventElapsedTime(&elapsedTime_cublas, start_cublas, stop_cublas));
  CUDA_CHECK(hipEventDestroy(start_cublas)); 
  CUDA_CHECK(hipEventDestroy(stop_cublas));
  elapsedTime_cublas = elapsedTime_cublas / n_rounds;
  float flops_cublas = getGFlops(elapsedTime_cublas, A.n_row, B.n_col, A.n_col);
  printf("kernel %-20s: %8.2f ms, %8.2f GFlops, %6.2f%% of cublas.\n", 
    "cublas", 
    elapsedTime_cublas, flops_cublas, 
    1 * 100.0);
  flops_info.push_back(flops_cublas);

  // copy data from device to host
  CUDA_CHECK(hipMemcpy(C.data, d_C.data, size_C * sizeof(float),  hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(d_A.data));
  CUDA_CHECK(hipFree(d_B.data));
  CUDA_CHECK(hipMemset(d_C.data, 0, size_C * sizeof(float)));
  CUDA_CHECK(hipFree(d_C.data));
  return;
}