
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void warpReduce() {
  int laneId = threadIdx.x & 0x1f;
  // Seed starting value as inverse lane ID
  //int value = 31 - laneId;
  int value = 1;
  // Use XOR mode to perform butterfly reduction
  for (int i=16; i>=1; i/=2)
    value += __shfl_xor_sync(0xffffffff, value, i, 32);
  // "value" now contains the sum across all threads
  printf("Thread %d final value = %d\n", threadIdx.x, value);
}
int main() {
  warpReduce<<< 1, 32 >>>();
  hipDeviceSynchronize();
  return 0;
}
