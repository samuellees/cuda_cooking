#include "hip/hip_runtime.h"
#include "conv_kernels.cuh"
#include "im2col.cuh"
#include "utils.cuh"
#include <iostream>

void conv_1x1_im2col_test() {
  // input size
  const int BATCH_SIZE = 32;
  const int Ci = 64;
  const int Hi = 64;
  const int Wi = 64;
  // kernel size
  const int Co = 64;
  const int Hk = 1; 
  const int Wk = 1;
  // padding, stride and dilation
  const int pad_h = 0;
  const int pad_w = 0;
  const int stride_h = 1;
  const int stride_w = 1;
  const int dilation_h = 1;
  const int dilation_w = 1;
  // output size
  const int Ho = Hi;
  const int Wo = Wi;
  // host data
  float* input = nullptr;
  float* kernel = nullptr;
  float* output = nullptr;
  const int size_input = BATCH_SIZE * Ci * Hi * Wi;
  const int size_kernel = Co * Ci * Hk * Wk;
  const int size_output = BATCH_SIZE * Co * Ho * Wo;
  malloc_and_init(&input, size_input);
  malloc_and_init(&kernel, size_kernel);
  malloc_and_init(&output, size_output);
  // device data
  float* d_input = nullptr;
  float* d_kernel = nullptr;
  float* d_output = nullptr;
  CUDA_CHECK(hipMalloc(&d_input, size_input * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_kernel, size_kernel * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_output, size_output * sizeof(float)));
  CUDA_CHECK(hipMemcpy(d_input, input, size_input * sizeof(float),  hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_kernel, kernel, size_kernel * sizeof(float),  hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_output, output, size_output * sizeof(float),  hipMemcpyHostToDevice));
  // prepare matrix size
  const int M = Co;
  const int K = Ci;
  const int N = Ho * Wo;


  // conv ref
  float* output_ref = nullptr;
  float* d_output_ref = nullptr;
  malloc_and_init(&output_ref, size_output);
  CUDA_CHECK(hipMalloc(&d_output_ref, size_output * sizeof(float)));
  CUDA_CHECK(hipMemcpy(d_output_ref, output_ref, size_output * sizeof(float),  hipMemcpyHostToDevice));
  float time_cudnn;
  convCuDNN(BATCH_SIZE, Ci, Hi, Wi, d_input,
            pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w, 
            Co, Hk, Wk, d_kernel, Ho, Wo, d_output_ref, &time_cudnn);
  time_cudnn = time_cudnn / n_rounds;
  printf("kernel %-20s: %8.2f ms, speedup=%.2f.\n", 
    "conv_cudnn", 
    time_cudnn, 
    1.0);
  

  // conv1x1
  float elapsedTime_conv_1x1;
  dim3 dims_block_conv_1x1(NTX, NTY);
  dim3 dims_grid_conv_1x1(CEIL_DIV(N, BLOCK_SIZE_L), BATCH_SIZE * CEIL_DIV(M, BLOCK_SIZE_L));
  // warm up
  kernel_conv_im2col_align<<<dims_grid_conv_1x1, dims_block_conv_1x1>>>(
    M, N, K, (float4*)d_kernel, (float4*)d_input, d_output);
  hipEvent_t start_conv_1x1, stop_conv_1x1;
  CUDA_CHECK(hipEventCreate(&start_conv_1x1)); 
  CUDA_CHECK(hipEventCreate(&stop_conv_1x1));
  CUDA_CHECK(hipEventRecord(start_conv_1x1, 0));
  for (int i = 0; i < n_rounds; ++i) {
    kernel_conv_im2col_align<<<dims_grid_conv_1x1, dims_block_conv_1x1>>>(
      M, N, K, (float4*)d_kernel, (float4*)d_input, d_output);
  }
  CUDA_CHECK(hipEventRecord(stop_conv_1x1, 0)); 
  CUDA_CHECK(hipEventSynchronize(stop_conv_1x1)); 
  CUDA_CHECK(hipEventElapsedTime(&elapsedTime_conv_1x1, start_conv_1x1, stop_conv_1x1));
  CUDA_CHECK(hipEventDestroy(start_conv_1x1)); 
  CUDA_CHECK(hipEventDestroy(stop_conv_1x1));
  elapsedTime_conv_1x1 = elapsedTime_conv_1x1 / n_rounds;
  printf("kernel %-20s: %8.2f ms, speedup=%.2f.\n", 
    "conv_1x1", 
    elapsedTime_conv_1x1, 
    elapsedTime_conv_1x1 / time_cudnn);


  // copy result to host
  CUDA_CHECK(hipMemcpy(output_ref, d_output_ref, size_output * sizeof(float),  hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(output, d_output, size_output * sizeof(float),  hipMemcpyDeviceToHost));
  // check
  std::cout << "check correctness..." << std::endl;
  bool error = false;
  #pragma unroll 64
  for (int i = 0; i < BATCH_SIZE * Co * Ho * Wo; ++i) {
    error = error || (std::abs(output[i] - output_ref[i]) > 1e-3);
  }
  std::cout << "error: " << error << std::endl;
  // free memory
  delete[] input;
  delete[] kernel;
  delete[] output;
  delete[] output_ref;
  hipFree(d_input);
  hipFree(d_kernel);
  hipFree(d_output);
  hipFree(d_output_ref);
}


void conv_NxN_im2col_with_batch_test() {
  // input size
  const int BATCH_SIZE = 1;
  const int Ci = 32;
  const int Hi = 32;
  const int Wi = 32;
  // kernel size
  const int Co = 32;
  const int Hk = 3;
  const int Wk = 3;
  // padding, stride and dilation
  const int pad_h = 0;
  const int pad_w = 0;
  const int stride_h = 2;
  const int stride_w = 2;
  const int dilation_h = 1;
  const int dilation_w = 1;
  // output size
  const int Ho = (Hi - ((Hk-1)*dilation_h+1) + 2*pad_h) / stride_h + 1;
  const int Wo = (Wi - ((Wk-1)*dilation_w+1) + 2*pad_w) / stride_w + 1;
  // prepare matrix size
  const int alignment = BLOCK_SIZE_L;
  // const int alignment = 8;
  const int M = Co;
  const int K = Ci * Hk * Wk;
  const int N = Ho * Wo;
  const int M_align = align(M, alignment);
  const int K_align = align(K, alignment);
  const int N_align = align(N, alignment);
  // column size
  const int Hc_align = K_align;
  const int Wc_align = N_align;
  // host data
  float* input = nullptr;
  float* kernel = nullptr;
  float* column = nullptr;
  float* column_align = nullptr;
  float* output = nullptr;
  float* output_align = nullptr;
  const int size_input = BATCH_SIZE * Ci * Hi * Wi;
  const int size_kernel = M * K;                    // (Co * Ci * Hk * Wk)
  const int size_kernel_align = M_align * K_align;  
  const int size_column = BATCH_SIZE * Hk * Wk * Ho * Wo * Ci;
  const int size_column_align = BATCH_SIZE * Hc_align * Wc_align;
  const int size_output = BATCH_SIZE * M * N; // = BATCH_SIZE * Co * Ho * Wo
  const int size_output_align = BATCH_SIZE * M_align * N_align;
  malloc_and_init(&input, size_input);
  malloc_and_init(&kernel, size_kernel);
  malloc_and_init(&column, size_column);
  malloc_and_init(&column_align, size_column_align);
  malloc_and_init(&output, size_output);
  malloc_and_init(&output_align, size_output_align);
  // device data
  float* d_input = nullptr;
  float* d_kernel = nullptr;
  float* d_kernel_align = nullptr;
  float* d_column = nullptr;
  float* d_column_align = nullptr;
  float* d_output = nullptr;
  float* d_output_align = nullptr;
  CUDA_CHECK(hipMalloc(&d_input, size_input * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_kernel, size_kernel * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_kernel_align, size_kernel_align * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_column, size_column * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_column_align, size_column_align * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_output, size_output * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_output_align, size_output_align * sizeof(float)));
  CUDA_CHECK(hipMemcpy(d_input, input, size_input * sizeof(float),  hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_kernel, kernel, size_kernel * sizeof(float),  hipMemcpyHostToDevice));

  // conv ref
  float* output_ref = nullptr;
  float* d_output_ref = nullptr;
  malloc_and_init(&output_ref, size_output);
  CUDA_CHECK(hipMalloc(&d_output_ref, size_output * sizeof(float)));
  CUDA_CHECK(hipMemcpy(d_output_ref, output_ref, size_output * sizeof(float),  hipMemcpyHostToDevice));
  float time_cudnn;
  convCuDNN(BATCH_SIZE, Ci, Hi, Wi, d_input,
            pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w, 
            Co, Hk, Wk, d_kernel, Ho, Wo, d_output_ref, &time_cudnn);
  time_cudnn = time_cudnn / n_rounds;
  printf("kernel %-20s: %8.2f ms, speedup=%.2f.\n", 
    "conv_cudnn", 
    time_cudnn, 
    1.0);
  

  // conv im2col
  float elapsedTime_conv;
  dim3 dims_block_conv(NTX, NTY);
  dim3 dims_grid_conv(CEIL_DIV(N, BLOCK_SIZE_L), BATCH_SIZE * CEIL_DIV(M, BLOCK_SIZE_L));
  // dim3 dims_block_conv(BLOCK_SIZE, BLOCK_SIZE);
  // dim3 dims_grid_conv(CEIL_DIV(N, BLOCK_SIZE_L), BATCH_SIZE * CEIL_DIV(M, BLOCK_SIZE_L));

  // im2col
  kernel_im2col_align<<<1024, 1024>>>(
    alignment,
    Ci, 
    Hi, Wi, 
    Ho, Wo,
    Hk, Wk, 
    stride_w, stride_h, 
    pad_w, pad_h, 
    dilation_w, dilation_h,
    d_input,
    d_column_align);
  // kernel_im2col<<<1024, 1024>>>(
  //   Ci, 
  //   Hi, Wi, 
  //   Ho, Wo,
  //   Hk, Wk, 
  //   stride_w, stride_h, 
  //   pad_w, pad_h, 
  //   dilation_w, dilation_h,
  //   d_input,
  //   d_column);
  
  // CUDA_CHECK(hipDeviceSynchronize());
  // CUDA_CHECK(hipMemcpy(column, d_column, size_column * sizeof(float),  hipMemcpyDeviceToHost));
  // CUDA_CHECK(hipMemcpy(column_align, d_column_align, size_column_align * sizeof(float),  hipMemcpyDeviceToHost));
  // CUDA_CHECK(hipDeviceSynchronize());

  // std::cout << "input1:" << std::endl;
  // print_matrix(input, Hi, Wi);

  // std::cout << "input2:" << std::endl;
  // print_matrix(input+Wi*Hi, Hi, Wi);

  // std::cout << "col:" << std::endl;
  // print_matrix(column, Ci*Hk*Wk, Wo*Ho);

  // std::cout << "col_align:" << std::endl;
  // print_matrix(column_align, Hc_align, Wc_align);

  // warm up
  kernel_conv_im2col_align<<<dims_grid_conv, dims_block_conv>>>(
    M_align, N_align, K_align, (float4*)d_kernel_align, (float4*)d_column_align, d_output_align);
  // align
  float time_padding = padding(d_kernel, d_kernel_align, M, K, M_align, K_align);
  hipEvent_t start_conv, stop_conv;
  CUDA_CHECK(hipEventCreate(&start_conv)); 
  CUDA_CHECK(hipEventCreate(&stop_conv));
  CUDA_CHECK(hipEventRecord(start_conv, 0));
  for (int i = 0; i < n_rounds; ++i) {
    kernel_conv_im2col_align<<<dims_grid_conv, dims_block_conv>>>(
      M_align, N_align, K_align, (float4*)d_kernel_align, (float4*)d_column_align, d_output_align);
  }
  CUDA_CHECK(hipEventRecord(stop_conv, 0)); 
  CUDA_CHECK(hipEventSynchronize(stop_conv)); 
  CUDA_CHECK(hipEventElapsedTime(&elapsedTime_conv, start_conv, stop_conv));
  CUDA_CHECK(hipEventDestroy(start_conv)); 
  CUDA_CHECK(hipEventDestroy(stop_conv));
  float time_unpadding = unpadding(d_output, d_output_align, M, N, M_align, N_align);
  elapsedTime_conv = elapsedTime_conv / n_rounds + time_unpadding + time_padding;
  printf("kernel %-20s: %8.2f ms, speedup=%.2f.\n", 
    "conv_NxN_im2col", 
    elapsedTime_conv, 
    elapsedTime_conv / time_cudnn);


  // copy result to host
  CUDA_CHECK(hipMemcpy(output_ref, d_output_ref, size_output * sizeof(float),  hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(output, d_output, size_output * sizeof(float),  hipMemcpyDeviceToHost));

  // std::cout << "output_ref:" << std::endl;
  // print_matrix(output_ref, M, N);

  // std::cout << "output:" << std::endl;
  // print_matrix(output, M, N);

  // check
  std::cout << "check correctness..." << std::endl;
  bool error = false;
  #pragma unroll 64
  for (int i = 0; i < BATCH_SIZE * Co * Ho * Wo; ++i) {
    error = error || (std::abs(output[i] - output_ref[i]) > 1e-3);
  }
  std::cout << "error: " << error << std::endl;
  // free memory
  free(input);
  free(kernel);
  free(output);
  free(output_ref);
  hipFree(d_input);
  hipFree(d_kernel);
  hipFree(d_column_align);
  hipFree(d_output);
  hipFree(d_output_align);
  hipFree(d_output_ref);
  free(column);
  hipFree(d_column);
  free(column_align);
}


void convCuDNN(
  const int BATCH_SIZE, const int Ci, const int Hi, const int Wi, const float* input,
  const int pad_h, const int pad_w, 
  const int stride_h, const int stride_w,
  const int dilation_h, const int dilation_w,
  const int Co, const int Hk, const int Wk, const float* kernel,
  const int Ho, const int Wo, float* output,
  float * time_ptr) {    
  //handle
  hipdnnHandle_t handle;
  hipdnnCreate(&handle);
  // tensor descriptor
  hipdnnTensorDescriptor_t input_desc;
  hipdnnTensorDescriptor_t output_desc;
  CUDNN_CHECK(hipdnnCreateTensorDescriptor(&input_desc));
  CUDNN_CHECK(hipdnnCreateTensorDescriptor(&output_desc));
  CUDNN_CHECK(hipdnnSetTensor4dDescriptor(
    input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    BATCH_SIZE, Ci, Hi, Wi));
  CUDNN_CHECK(hipdnnSetTensor4dDescriptor(
    output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    BATCH_SIZE, Co, Ho, Wo));
  // kernel 
  hipdnnFilterDescriptor_t kernel_desc;
  CUDNN_CHECK(hipdnnCreateFilterDescriptor(&kernel_desc));
  CUDNN_CHECK(hipdnnSetFilter4dDescriptor(
    kernel_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    Co, Ci, Hk, Wk));
  // convolution descriptor
  hipdnnConvolutionDescriptor_t conv_desc;
  hipdnnCreateConvolutionDescriptor(&conv_desc);
  CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(conv_desc,
    pad_h, pad_w,           // padding
    stride_h, stride_w,     // stride
    dilation_h, dilation_w, // dilation
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
  // algorithm
  hipdnnConvolutionFwdAlgo_t algo;
  CUDNN_CHECK(hipdnnGetConvolutionForwardAlgorithm(
    handle, input_desc, kernel_desc, conv_desc, output_desc, 
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
    0, &algo));
  // workspace size && allocate memory
  size_t workspace_size = 0;
  CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(handle,
    input_desc, kernel_desc, conv_desc, output_desc,
    algo, &workspace_size));
  void * workspace = nullptr;
  CUDA_CHECK(hipMalloc(&workspace, workspace_size));
  // convolution
  auto alpha = 1.0f, beta = 0.0f;
  // warm  up
  CUDNN_CHECK(hipdnnConvolutionForward(handle,
    &alpha, input_desc, input,
    kernel_desc, kernel,
    conv_desc, algo,
    workspace, workspace_size,
    &beta, output_desc, output));
  hipEvent_t start_conv_ref, stop_conv_ref;
  CUDA_CHECK(hipEventCreate(&start_conv_ref)); 
  CUDA_CHECK(hipEventCreate(&stop_conv_ref));
  CUDA_CHECK(hipEventRecord(start_conv_ref, 0));
  for (int i = 0; i < n_rounds; ++i) {
    CUDNN_CHECK(hipdnnConvolutionForward(handle,
      &alpha, input_desc, input,
      kernel_desc, kernel,
      conv_desc, algo,
      workspace, workspace_size,
      &beta, output_desc, output));
  }
  CUDA_CHECK(hipEventRecord(stop_conv_ref, 0)); 
  CUDA_CHECK(hipEventSynchronize(stop_conv_ref)); 
  CUDA_CHECK(hipEventElapsedTime(time_ptr, start_conv_ref, stop_conv_ref));
  CUDA_CHECK(hipEventDestroy(start_conv_ref)); 
  CUDA_CHECK(hipEventDestroy(stop_conv_ref));
  // destroy
  hipFree(workspace);
  hipdnnDestroyTensorDescriptor(input_desc);
  hipdnnDestroyTensorDescriptor(output_desc);
  hipdnnDestroyConvolutionDescriptor(conv_desc);
  hipdnnDestroyFilterDescriptor(kernel_desc);
  hipdnnDestroy(handle);
} 